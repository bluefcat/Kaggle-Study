
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void thread_id(){
    printf("---inThread---\n[%d %d %d]\n[%d %d %d] %u\n---out---\n", 
    blockIdx.x, blockIdx.y, blockIdx.z, 
    threadIdx.x, threadIdx.y, threadIdx.z, clock());
    return;
}

int main(){
    dim3 threadsPerBlock(3, 3, 3);

    thread_id<<<2, threadsPerBlock>>>();
    getchar();
    return 0;
}